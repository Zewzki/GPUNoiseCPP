#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <SFML/Window.hpp>
#include <SFML/Graphics.hpp>

#define WIDTH 800
#define HEIGHT 600
#define ARR_SIZE 512

double F2 = .5 * (sqrt(3.0) - 1.0);
double G2 = (3.0 - sqrt(3.0)) / 6.0;

__constant__ int iterations = 8;
__constant__ double scale = 0.01;
__constant__ double persistance = 0.6;
__constant__ int high = 255;
__constant__ int low = 0;
__constant__ int w = WIDTH;
__constant__ int h = HEIGHT;

__constant__ int perm[ARR_SIZE];
__constant__ int permMod12[ARR_SIZE];
const int p[] = {151, 160, 137, 91, 90, 15, 131, 13, 201, 95, 96, 53, 194, 233, 7, 225, 140, 36, 103, 30, 69, 142, 8, 99, 37, 240, 21, 10, 23, 190, 6, 148, 247, 120, 234, 75, 0, 26, 197, 62, 94, 252, 219, 203, 117, 35, 11, 32, 57, 177, 33, 88, 237, 149, 56, 87, 174, 20, 125, 136, 171, 168, 68, 175, 74, 165, 71, 134, 139, 48, 27, 166, 77, 146, 158, 231, 83, 111, 229, 122, 60, 211, 133, 230, 220, 105, 92, 41, 55, 46, 245, 40, 244, 102, 143, 54, 65, 25, 63, 161, 1, 216, 80, 73, 209, 76, 132, 187, 208, 89, 18, 169, 200, 196, 135, 130, 116, 188, 159, 86, 164, 100, 109, 198, 173, 186, 3, 64, 52, 217, 226, 250, 124, 123, 5, 202, 38, 147, 118, 126, 255, 82, 85, 212, 207, 206, 59, 227, 47, 16, 58, 17, 182, 189, 28, 42, 223, 183, 170, 213, 119, 248, 152, 2, 44, 154, 163, 70, 221, 153, 101, 155, 167, 43, 172, 9, 129, 22, 39, 253, 19, 98, 108, 110, 79, 113, 224, 232, 178, 185, 112, 104, 218, 246, 97, 228, 251, 34, 242, 193, 238, 210, 144, 12, 191, 179, 162, 241, 81, 51, 145, 235, 249, 14, 239, 107, 49, 192, 214, 31, 181, 199, 106, 157, 184, 84, 204, 176, 115, 121, 50, 45, 127, 4, 150, 254, 138, 236, 205, 93, 222, 114, 67, 29, 24, 72, 243, 141, 128, 195, 78, 66, 215, 61, 156, 180 };

__constant__ int grad3[][3] = { { 1, 1, 0 }, { -1, 1, 0 }, { 1, -1, 0 }, { -1, -1, 0 }, { 1, 0, 1 }, { -1, 0, 1 }, { 1, 0, -1 }, { -1, 0, -1 }, { 0, 1, 1 }, { 0, -1, 1 }, { 0, 1, -1 }, { 0, -1, -1 } };

__device__ double mix(double a, double b, double t) {
  return (1 - t) * a + t * b;
}

__device__ double fade(double t) {
  return t * t * t * (t * (t * 6 - 15) + 10);
}

__device__ double dot(int g[], double x, double y, double z) {
  return g[0] * x + g[1] * y + g[2] * z;
}

__device__ int fastFloor(double x) {
  return x > 0 ? (int) x : (int) x - 1;
}

__device__ double noise(double x, double y, double z) {

  int X = fastFloor(x);
  int Y = fastFloor(y);
  int Z = fastFloor(z);

  x = x - X;
  y = y - Y;
  z = z - Z;

  X = X & 255;
  Y = Y & 255;
  Z = Z & 255;

  int gi000 = perm[X + perm[Y + perm[Z]]] % 12;
  int gi001 = perm[X + perm[Y + perm[Z + 1]]] % 12;
  int gi010 = perm[X + perm[Y + 1 + perm[Z]]] % 12;
  int gi011 = perm[X + perm[Y + 1 + perm[Z + 1]]] % 12;
  int gi100 = perm[X + 1 + perm[Y + perm[Z]]] % 12;
  int gi101 = perm[X + 1 + perm[Y + perm[Z + 1]]] % 12;
  int gi110 = perm[X + 1 + perm[Y + 1 + perm[Z]]] % 12;
  int gi111 = perm[X + 1 + perm[Y + 1 + perm[Z + 1]]] % 12;

  double n000 = dot(grad3[gi000], x, y, z);
  double n100 = dot(grad3[gi100], x - 1, y, z);
  double n010 = dot(grad3[gi010], x, y - 1, z);
  double n110 = dot(grad3[gi110], x - 1, y - 1, z);
  double n001 = dot(grad3[gi001], x, y, z - 1);
  double n101 = dot(grad3[gi101], x - 1, y, z - 1);
  double n011 = dot(grad3[gi011], x, y - 1, z - 1);
  double n111 = dot(grad3[gi111], x - 1, y - 1, z - 1);

  double u = fade(x);
  double v = fade(y);
  double w = fade(z);

  double nx00 = mix(n000, n100, u);
  double nx01 = mix(n001, n101, u);
  double nx10 = mix(n010, n110, u);
  double nx11 = mix(n011, n111, u);

  double nxy0 = mix(nx00, nx10, v);
  double nxy1 = mix(nx01, nx11, v);
  double nxyz = mix(nxy0, nxy1, w);

  return nxyz;

}

__global__ void sumOctave(int z, sf::Uint8 *result) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  int x = i % blockDim.x;
  int y = (int) (i / blockDim.x);
  
  double maxAmp = 0.0;
  double amp = 1.0;
  double freq = scale;
  double n = 0;
  
  for(int i = 0; i < iterations; i++) {
  
    double adding = noise(x * freq, y * freq, z * freq) * amp;
    n += adding;
    maxAmp += amp;
    amp *= persistance;
    freq *= 2;
  
  }
  
  n /= maxAmp;
  
  n = n * (high - low) / 2 + (high + low) / 2;
  
  result[(i * 4)] = n;
  result[(i * 4) + 1] = n;
  result[(i * 4) + 2] = n;
  //result[(i * 4) + 3] = 255;
  
  //result[i] = n;
  
  //printf("(%d, %d) = %d", x, y, n);

}

int main(void) {

  int N = WIDTH * HEIGHT;
  
  int blockSize = 512;
  int nBlocks = (N / blockSize) + 1;
  
  int * hostPerm;
  int * hostPermMod12;
  
  hostPerm = new int[ARR_SIZE];
  hostPermMod12 = new int[ARR_SIZE];
  
  for(int i = 0; i < ARR_SIZE; i++) {
    hostPerm[i] = p[i & 255];
    hostPermMod12[i] = (hostPerm[i] % 12);
  }
  
  hipMemcpyToSymbol(HIP_SYMBOL(perm), hostPerm, ARR_SIZE * sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(permMod12), hostPermMod12, ARR_SIZE * sizeof(int));

  // host array
  sf::Uint8 *screen = (sf::Uint8*) malloc(N * 4 * sizeof(sf::Uint8));
  for(int i = 0; i < N * 4; i++) screen[i] = 255;
  
  // device array
  sf::Uint8 *d_screen = new sf::Uint8[WIDTH * HEIGHT * 4];
  hipMalloc(&d_screen, N * 4 * sizeof(sf::Uint8));
  
  sf::Texture texture;
  if (!texture.create(WIDTH, HEIGHT)) return -1;
  
  sf::Sprite sprite(texture);
  
  int z = 0;
  
  sf::RenderWindow window(sf::VideoMode(WIDTH, HEIGHT), "Noise");
  
  hipMemcpy(d_screen, screen, N * 4 * sizeof(sf::Uint8), hipMemcpyHostToDevice);
  
  //window.setVerticalSyncEnabled(true);
  
  while(window.isOpen()) {
  
    sf::Event event;
    
    // close window when 'x' is pressed, thus exiting outer loop
    while(window.pollEvent(event)) if (event.type == sf::Event::Closed) window.close();
    
    // call kernel, sync, and copy info off of gpu
    //sumOctave<<<nBlocks, blockSize>>>(z, d_screen);
    sumOctave<<<600, 800>>>(z, d_screen);
    hipDeviceSynchronize();
    hipMemcpy(screen, d_screen, N * 4 * sizeof(sf::Uint8), hipMemcpyDeviceToHost);
    
    // update texture
    texture.update(screen);
    window.draw(sprite);
 
    window.display();
    
    z++;
  
  }
  
  hipFree(d_screen);
  free(screen);
  
  return 0;
  
}
